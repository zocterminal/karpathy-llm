#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net trained in raw CUDA
Non-trivial notes to be aware of:

We are being clever in the backward pass to conserve memory.
In particular, all parameters use a += in the backward pass, so we
can later do gradient accumulation. But all activations have = instead of +=
because these are faster (just read, no write). This is okay for all activations
except for those in the residual stream, where the gradients have to add. We make
sure that those parts work out ok and that we do a += as necessary. E.g.,
the layernorms are connected to the residuals so we += in layernorm backward.

In this file we are using Mixed Precision training, so different activations,
paramaters, grads and buffers may be kept at different precisions, to take
advantage of the fast low-precision hardware in the latest GPUs (bf16/fp16),
and fp8 (coming soon^TM).

Compile:
make train_gpt2cu

Example launch using bfloat16 on 1 GPU batch size 8, sample/eval every 200 steps:
Also we're using TinyStories here for example as it is a bigger dataset
./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories

Example launch using bfloat16 on 4 GPUs, same as above:
mpirun -np 4 ./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <unistd.h>
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#ifdef MULTI_GPU
#include <mpi.h>
#include <nccl.h>
#endif

// ----------------------------------------------------------------------------
// CUDA precision settings

// turn on bf16 as default, done up here for now
#define ENABLE_BF16

// use bf16 (bfloat 16)
#if defined(ENABLE_BF16)
typedef __hip_bfloat16 floatX;
typedef float floatN;
#define CUBLAS_LOWP HIP_R_16BF
#define CUBLAS_LOWP_COMPUTE HIPBLAS_COMPUTE_32F

#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclBfloat16;
const ncclDataType_t ncclFloatN = ncclFloat;
#endif

// use fp16 (note: this may require gradient scaler, currently not implemented!)
#elif defined(ENABLE_FP16)
typedef half floatX;
typedef float floatN;
#define CUBLAS_LOWP HIP_R_16F
#define CUBLAS_LOWP_COMPUTE HIPBLAS_COMPUTE_32F

#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclHalf;
const ncclDataType_t ncclFloatN = ncclFloat;
#endif

// fallback for fp32
#else
typedef float floatX;
typedef float floatN;
#define CUBLAS_LOWP HIP_R_32F
#define CUBLAS_LOWP_COMPUTE cublas_compute_type // auto-select FP32 vs TF32

#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclFloat;
const ncclDataType_t ncclFloatN = ncclFloat;
#endif

#endif

// ----------------------------------------------------------------------------
// CUDA utils

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
hipblasHandle_t cublas_handle;
hipblasLtHandle_t cublaslt_handle;

namespace cg = cooperative_groups;

// convenience macro for calculating grid/block dimensions for kernels
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// CUDA error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// cuBLAS error checking
void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }

#ifdef MULTI_GPU
void nccl_check(ncclResult_t status, const char *file, int line) {
    if (status != ncclSuccess) {
        printf("[NCCL ERROR] at file %s:%d:\n%s\n", file, line, ncclGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}
#define ncclCheck(err) (nccl_check(err, __FILE__, __LINE__))

void mpi_check(int status, const char *file, int line) {
    if (status != MPI_SUCCESS) {
        char mpi_error[4096];
        int mpi_error_len = 0;
        assert(MPI_Error_string(status, &mpi_error[0], &mpi_error_len) == MPI_SUCCESS);
        printf("[MPI ERROR] at file %s:%d:\n%.*s\n", file, line, mpi_error_len, mpi_error);
        exit(EXIT_FAILURE);
    }
}
#define mpiCheck(err) (mpi_check(err, __FILE__, __LINE__))
#endif

// GPU helper functions for atomicAdd on smaller than 32-bit types
#ifdef ENABLE_BF16
__device__ void atomicAddX(__hip_bfloat16* addr, __hip_bfloat16 val) {
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    __hip_bfloat162* ptr_bf16 = reinterpret_cast<__hip_bfloat162*>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    __hip_bfloat162 add_val = (ptr_val & 0x3) ? __halves2bfloat162(__ushort_as_bfloat16(0), val)
                                             : __halves2bfloat162(val, __ushort_as_bfloat16(0));
    atomicAdd(ptr_bf16, add_val);
}
#endif

#ifdef ENABLE_FP16
__device__ void atomicAddX(half* addr, half val) {
    uintptr_t ptr_val = reinterpret_cast<uintptr_t>(addr);
    half2* ptr_fp16 = reinterpret_cast<half2*>(ptr_val & ~uintptr_t(0x3));

    // Prepare the value to add, setting the other half to zero
    half2 add_val = (ptr_val & 0x3) ? __halves2half2(__ushort_as_half(0), val)
                                    : __halves2half2(val, __ushort_as_half(0));
    atomicAdd(ptr_fp16, add_val);
}
#endif

__device__ void atomicAddX(float* addr, float val) {
    atomicAdd(addr, val);
}

// ----------------------------------------------------------------------------
// Random Number Generatiom

// Simple xorshift RNG
__device__ __host__ unsigned int random_u32(unsigned long long *state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
__device__ __host__ float random_f32(unsigned long long *state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}

// SquirrelNoise5 - Squirrel's Raw Noise utilities (version 5)
// This gives us a random number from threadIdx/blockIdx + a single seed for the entire GPU
// todo - possibly overkill and we don't need such high quality random numbers? (tbd)
// http://eiserloh.net/noise/SquirrelNoise5.hpp
__device__ __host__ constexpr unsigned int SquirrelNoise5(int positionX, unsigned int seed)
{
	constexpr unsigned int SQ5_BIT_NOISE1 = 0xd2a80a3f;	// 11010010101010000000101000111111
	constexpr unsigned int SQ5_BIT_NOISE2 = 0xa884f197;	// 10101000100001001111000110010111
	constexpr unsigned int SQ5_BIT_NOISE3 = 0x6C736F4B; // 01101100011100110110111101001011
	constexpr unsigned int SQ5_BIT_NOISE4 = 0xB79F3ABB;	// 10110111100111110011101010111011
	constexpr unsigned int SQ5_BIT_NOISE5 = 0x1b56c4f5;	// 00011011010101101100010011110101
	unsigned int mangledBits = (unsigned int) positionX;
	mangledBits *= SQ5_BIT_NOISE1;
	mangledBits += seed;
	mangledBits ^= (mangledBits >> 9);
	mangledBits += SQ5_BIT_NOISE2;
	mangledBits ^= (mangledBits >> 11);
	mangledBits *= SQ5_BIT_NOISE3;
	mangledBits ^= (mangledBits >> 13);
	mangledBits += SQ5_BIT_NOISE4;
	mangledBits ^= (mangledBits >> 15);
	mangledBits *= SQ5_BIT_NOISE5;
	mangledBits ^= (mangledBits >> 17);
	return mangledBits;
}
__device__ __host__ constexpr unsigned int Get1dNoiseUint(int positionX, unsigned int seed)
{
	return SquirrelNoise5(positionX, seed);
}
__device__ __host__ constexpr unsigned int Get2dNoiseUint(int indexX, int indexY, unsigned int seed)
{
	constexpr int PRIME_NUMBER = 198491317; // Large prime number with non-boring bits
	return SquirrelNoise5(indexX + (PRIME_NUMBER * indexY), seed);
}
__device__ __host__ constexpr float Get1dNoiseZeroToOne(int index, unsigned int seed)
{
	constexpr double ONE_OVER_MAX_UINT = (1.0 / (double) 0xFFFFFFFF);
	return (float)(ONE_OVER_MAX_UINT * (double) SquirrelNoise5(index, seed));
}
__device__ __host__ constexpr float Get2dNoiseZeroToOne(int indexX, int indexY, unsigned int seed)
{
	constexpr double ONE_OVER_MAX_UINT = (1.0 / (double) 0xFFFFFFFF);
	return (float)(ONE_OVER_MAX_UINT * (double) Get2dNoiseUint(indexX, indexY, seed));
}

// stochastic rounding built on top of Squirel Noise above (with seed updated per step via xorshift)
__device__ __forceinline__ void stochastic_rounding(float in, __hip_bfloat16 *out, unsigned int seed) {
    // todo - is this stochastic rounding *too good*? can we cut any corners?
    unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
    unsigned int threshold = random & 0xFFFF;
    unsigned int float_bits = __float_as_uint(in);
    unsigned int rounded_bits = float_bits & 0x0000FFFF;
    float_bits = (rounded_bits > threshold) ? (float_bits | 0xFFFF) : (float_bits  & ~0xFFFF);
    *out = __float2bfloat16_rn(__uint_as_float(float_bits));
}
__device__ __forceinline__ void stochastic_rounding(float in, half *out, unsigned int random) {
    *out = (float)in; // todo - implement this...
}
__device__ __forceinline__ void stochastic_rounding(float in, float *out, unsigned int random) {
    *out = in; // dummy function for when floatX is float (FP32 mode)
}

// ----------------------------------------------------------------------------
// fread convenience utils, with nice handling of error checking using macros
// simple replace fopen, fread, fclose with fopenCheck, freadCheck, fcloseCheck

FILE *fopen_check(const char *path, const char *mode, const char *file, int line) {
    FILE *fp = fopen(path, mode);
    if (fp == NULL) {
        fprintf(stderr, "Error: Failed to open file '%s' at %s:%d\n", path, file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Path: %s\n", path);
        fprintf(stderr, "  Mode: %s\n", mode);
        exit(EXIT_FAILURE);
    }
    return fp;
}

#define fopenCheck(path, mode) fopen_check(path, mode, __FILE__, __LINE__)

void fread_check(void *ptr, size_t size, size_t nmemb, FILE *stream, const char *file, int line) {
    size_t result = fread(ptr, size, nmemb, stream);
    if (result != nmemb) {
        if (feof(stream)) {
            fprintf(stderr, "Error: Unexpected end of file at %s:%d\n", file, line);
        } else if (ferror(stream)) {
            fprintf(stderr, "Error: File read error at %s:%d\n", file, line);
        } else {
            fprintf(stderr, "Error: Partial read at %s:%d. Expected %zu elements, read %zu\n",
                    file, line, nmemb, result);
        }
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Expected elements: %zu\n", nmemb);
        fprintf(stderr, "  Read elements: %zu\n", result);
        exit(EXIT_FAILURE);
    }
}

#define freadCheck(ptr, size, nmemb, stream) fread_check(ptr, size, nmemb, stream, __FILE__, __LINE__)

void fclose_check(FILE *fp, const char *file, int line) {
    if (fclose(fp) != 0) {
        fprintf(stderr, "Error: Failed to close file at %s:%d\n", file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        exit(EXIT_FAILURE);
    }
}

#define fcloseCheck(fp) fclose_check(fp, __FILE__, __LINE__)

// ----------------------------------------------------------------------------
// malloc error-handling wrapper util

void *malloc_check(size_t size, const char *file, int line) {
    void *ptr = malloc(size);
    if (ptr == NULL) {
        fprintf(stderr, "Error: Memory allocation failed at %s:%d\n", file, line);
        fprintf(stderr, "Error details:\n");
        fprintf(stderr, "  File: %s\n", file);
        fprintf(stderr, "  Line: %d\n", line);
        fprintf(stderr, "  Size: %zu bytes\n", size);
        exit(EXIT_FAILURE);
    }
    return ptr;
}

#define mallocCheck(size) malloc_check(size, __FILE__, __LINE__)

// ----------------------------------------------------------------------------
// MPI / multi-processing setup

// Parameters specific to training on multiple GPUs.
typedef struct {
    int process_rank;      // Rank of this process among all MPI processes. 0 if no multi-GPU.
    int num_processes;     // Total number of processes. 1 if no multi-GPU.
    int local_device_idx;  // This process GPU index on current machine. 0 if no multi-GPU.
#ifdef MULTI_GPU
    ncclComm_t nccl_comm;  // NCCL communication primitive, used for collective mutli-GPU work.
#endif
} MultiGpuConfig;

// one global variable to hold the multi-GPU configuration for this process
MultiGpuConfig multi_gpu_config;

#ifdef MULTI_GPU
// Determine which GPU this process should use.
// Processes on the same machines use different GPU indicies. Processes on other machines don't.
// Copied from NCCL examples: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-2-one-device-per-process-or-thread
int multi_gpu_get_local_device_idx(int process_rank, int num_processes) {
  char hostname[1024];
  hostname[1023] = '\0';
  // All processes on the same machine will share the same hostname.
  gethostname(hostname, 1023);
  for (int i=0; i < 1024; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        break;
    }
  }
  uint64_t hostname_hash = 5381;
  for (int c = 0; hostname[c] != '\0'; c++){ hostname_hash = ((hostname_hash << 5) + hostname_hash) ^ hostname[c]; }

  // Distribute all hostname hashes to all processes.
  uint64_t* all_hostsname_hashes = (uint64_t*)malloc(num_processes * sizeof(uint64_t));
  all_hostsname_hashes[process_rank] = hostname_hash;
  mpiCheck(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, all_hostsname_hashes, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Identify which GPU we need to use.
  int local_device_idx = 0;
  for (int current_process = 0; current_process < num_processes; ++current_process) {
     if (current_process == process_rank) {
      // Found my gpu, local_device_idx now has my target GPU index.
      break;
     }
     if (all_hostsname_hashes[current_process] == all_hostsname_hashes[process_rank]) {
      // This process ID runs on the same machine, but it's not me, skip this GPU
      local_device_idx++;
     }
  }

  free(all_hostsname_hashes);
  return local_device_idx;
}
#endif

MultiGpuConfig multi_gpu_config_init(int *argc, char ***argv) {
#ifdef MULTI_GPU
    // Initialize MPI.
    MultiGpuConfig result;
    mpiCheck(MPI_Init(argc, argv));
    mpiCheck(MPI_Comm_rank(MPI_COMM_WORLD, &result.process_rank));
    mpiCheck(MPI_Comm_size(MPI_COMM_WORLD, &result.num_processes));
    result.local_device_idx = multi_gpu_get_local_device_idx(result.process_rank, result.num_processes);
    cudaCheck(hipSetDevice(result.local_device_idx));
    ncclUniqueId nccl_id;
    if (result.process_rank == 0) {
        ncclCheck(ncclGetUniqueId(&nccl_id));
    }
    mpiCheck(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));
    ncclCheck(ncclCommInitRank(&result.nccl_comm, result.num_processes, nccl_id, result.process_rank));
    return result;
#else
    printf("Multi-GPU support is disabled. Using a single GPU.");
    return MultiGpuConfig{
        .process_rank = 0,
        .num_processes = 1,
        .local_device_idx = 0,
    };
#endif
}

void multi_gpu_config_free(const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    ncclCheck(ncclCommDestroy(multi_gpu_config->nccl_comm));
    mpiCheck(MPI_Finalize());
#endif
}

// convenience function that only prints if the rank of process is zero
void printf0(const char *format, ...) {
    if (multi_gpu_config.process_rank == 0) {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
    }
}

// ----------------------------------------------------------------------------
// all the kernels

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

template <typename TOut, typename Tw>
__global__ void encoder_forward_kernel2(TOut* out,
                               int* inp, Tw* wte, Tw* wpe,
                               int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        TOut* out_btc = out + b * T * C + t * C + c;
        Tw* wte_ix = wte + ix * C + c;
        Tw* wpe_tc = wpe + t * C + c;
        *out_btc = (TOut)((float)*wte_ix + (float)*wpe_tc);
    }
}

// really bad naive kernel with atomicAdd
template <typename Type, typename Tdout>
__global__ void encoder_backward_kernel(Type* dwte, Type* dwpe,
                                        const Tdout* dout, const int* inp,
                                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        const Tdout* dout_btc = dout + b * T * C + t * C + c;
        Type* dwte_ix = dwte + ix * C + c;
        Type* dwpe_tc = dwpe + t * C + c;

        atomicAddX(dwte_ix, (Type)*dout_btc);
        atomicAddX(dwpe_tc, (Type)*dout_btc);
    }
}

// currently reads FP32, outputs floatX(FP16/BF16/FP8)
template <typename Type, typename TOut, typename TParam>
__global__ void layernorm_forward_kernel3(TOut* __restrict__ out, Type* __restrict__ mean, Type* __restrict__ rstd,
                                    const Type*  __restrict__ inp, const TParam*  __restrict__ weight,
                                    const TParam* __restrict__ bias, int N, int C) {
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) { return; } // guard

    // the row of input that this group of threads is responsible for
    const Type* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += (float)x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, (Type)m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = (float)x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, (Type)s);
    }

    // final normalization and scaling by weight/bias
    TOut* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * ((float)__ldcs(x+c) - m);
        __stcs(o+c, (TOut)(n * (float)weight[c] + (float)bias[c]));
    }
}

// inputs floatX, outputs FP32 (for current FP32-only activation path for this WIP)
__global__ void permute_kernel(floatX* q, floatX* k, floatX* v,
                               const floatX* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        q[idx] = __ldcs(&inp[inp_idx]);
        k[idx] = __ldcs(&inp[inp_idx + NH * d]);
        v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
    }
}

__global__ void permute_kernel_backward(floatX* dinp,
                                        const floatX* dq, const floatX* dk, const floatX* dv,
                                        int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        dinp[inp_idx] = dq[idx];
        dinp[inp_idx + NH * d] = dk[idx];
        dinp[inp_idx + 2 * (NH * d)] = dv[idx];
    }
}

__global__ void unpermute_kernel(floatX* inp, floatX *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = __ldcs(&inp[idx]);
    }
}

__global__ void unpermute_kernel_backward(floatX* dinp, const floatX *dout, int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        dinp[idx] = (floatX)dout[other_idx];
    }
}

template <typename Type>
__global__ void softmax_forward_kernel5(Type* out, float inv_temperature, const Type* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // micro-optimization: we iterate backwards so that
    // after the softmax backward operation completes, the cache retains the
    // part of the matrix close to the upper left corner, which benefits the
    // matmul operation that immediately follows.
    // int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank(); // forward order
    int idx = (gridDim.x - blockIdx.x - 1) * warp.meta_group_size() + warp.meta_group_rank(); // backward order
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const Type* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    const Type* x_aligned = reinterpret_cast<const Type*>(__builtin_assume_aligned(x, 16));
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float regarray[4];
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            regarray[k] = (float)x_aligned[4*i + k];
        }
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, regarray[k]);
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (regarray[k] - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, (float)x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * ((float)x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * ((float)__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, (Type)(ev * norm));
    }
}

template <typename TOut, typename T1, typename T2>
__global__ void residual_forward_kernel(TOut* out, T1* inp1, T2* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = (TOut)((float)__ldcs(&inp1[idx]) + (float)__ldcs(&inp2[idx]));
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
__global__ void gelu_forward_kernel(floatX* out, const floatX* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = (float)inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = (floatX)(0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube))));
    }
}

__global__ void gelu_backward_kernel(floatX* dinp, const floatX* inp, const floatX* dout, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float x = (float)inp[i];
        float cube = 0.044715f * x * x * x;
        float tanh_arg = GELU_SCALING_FACTOR * (x + cube);
        float tanh_out = tanhf(tanh_arg);
        float coshf_out = coshf(tanh_arg);
        float sech_out = 1.0f / (coshf_out * coshf_out);
        float local_grad = 0.5f * (1.0f + tanh_out) + x * 0.5f * sech_out * GELU_SCALING_FACTOR * (1.0f + 3.0f * 0.044715f * x * x);
        dinp[i] = (floatX)(local_grad * (float)dout[i]);
    }
}

__global__ void softmax_forward_kernel7(float* out, const float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel4, but optimised for very large Cs with advanced unrolling

    // The trick is to read into a register array (all indices known at compile time)
    // and always read UNROLL_FACTOR values to maximise memory level parallelism
    // even if we would be out of bounds, we set the index to min(C-1, idx)
    // so we just do some unnecessary reads (obviously bad for small C)
    // the writes are in a separate loop with a conditional check for out of bounds
    // making it separate is necessary to convince the compiler to do the right thing
    const int UNROLL_FACTOR = 8;
    const int warpsPerBlock = blockDim.x / 32;

    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    if (tid >= C) {
        maxvals[warpId] = -INFINITY;
        sumvals[warpId] = 0.0f;
        return;
    }

    const float* x = inp + idx * C; // input
    float* y = out + idx * C; // output

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x * UNROLL_FACTOR) {
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            maxval = fmaxf(maxval, x[min(C - 1, i + u*blockDim.x)]);
        }
    }

    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);
    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();
    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        #pragma unroll
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    // + thread coarsening for sum
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x * UNROLL_FACTOR) {
        float reg_array[UNROLL_FACTOR];
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            reg_array[u] = __ldcs(&x[min(C - 1, i + u*blockDim.x)]);
        }
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            if (i + u*blockDim.x < C) {
                float output = expf(reg_array[u] - offset);
                y[min(C - 1, i + u*blockDim.x)] = output; // compiler likes redundant min()?!
                sumval += output; // combined into the same loop unlike kernel3
            }
        }
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);
    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();
    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        #pragma unroll
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x * UNROLL_FACTOR) {
        float reg_array[UNROLL_FACTOR];
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            reg_array[u] = y[min(C - 1, i + u*blockDim.x)];
        }
        #pragma unroll
        for (int u = 0; u < UNROLL_FACTOR; u++) {
            if (i + u*blockDim.x < C) {
                y[i + u*blockDim.x] = reg_array[u] / sum;
            }
        }
    }
}

// this kernel performs a column-wise reduction over dout, in PyTorch equivalent to:
// dbias = dout.sum((0,1))
// the idea is to employ one block to reduce along several columns,
// where each block has a width of 32 columns to ensure coalesced access.
// at the end we accumulate the reductions performed by the warps in each block via shared memory
template <typename Td>
__global__ void matmul_backward_bias_kernel4(Td* dbias, const Td* dout, int B, int T, int OC) {
    // this kernel is launched with 1D grid_dim of OC/32
    // for example let's say block_size is 128
    extern __shared__ float smem[]; // of size block_size (128)
    const int warp_id = threadIdx.x / warpSize; // warp index in the block, 0,1,2,3
    const int lane_id = threadIdx.x % warpSize; // thread index in the warp, 0,1,2,...,31
    const int tl = blockIdx.x * warpSize; // pointer to the start column for this block
    const int vstep = blockDim.x / warpSize; // number of warps in a block, e.g. 4

    // pointer to the start of the column for one lane of threads
    // so e.g. 4 threads (of the same lane_id) will reduce this one column
    const Td* dout_col = dout + tl + lane_id;

    // column reductions by looping through the rows
    // each of the 4 threads offsets by its warp_id and then skips by vstep
    // together these 4 threads cover all B*T rows of this (lane_id) column
    // importantly, consecutive threads (in threadId) are processing adjacent columns,
    // leading to a coalesced memory access pattern
    float dout_sum = 0.0f;
    for (int row = warp_id; row < B * T; row += vstep) {
        dout_sum += (float)dout_col[row * OC];
    }
    smem[lane_id + warp_id * warpSize] = dout_sum;
    __syncthreads();

    // warp_id 0 reduces the shared memory column-wise, linearly
    dout_sum = 0.0f;
    if (warp_id == 0) {
        for (int j = 0; j < vstep; j++) {
            dout_sum += smem[lane_id + j * warpSize];
        }
        dbias[tl + lane_id] = (Td)dout_sum;
    }
}

// uses shared memory instead for the reduces
template <typename Tdinp, typename Tparams, typename Tdout, typename Trest>
__global__ void layernorm_backward_kernel2(Tdinp* dinp, Tparams* dweight, Tparams* dbias,
                        const Tdout* dout, const Trest* inp, const Tparams* weight, const Trest* mean, const Trest* rstd,
                        int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    int N = B * T;
    if(idx >= N) { return; } // thread guards

    int b = idx / T;
    int t = idx % T;

    const Tdout* dout_bt = dout + b * T * C + t * C;
    const Trest* inp_bt = inp + b * T * C + t * C;
    Tdinp* dinp_bt = dinp + b * T * C + t * C;
    const float mean_bt = (float)mean[b * T + t];
    const float rstd_bt = (float)rstd[b * T + t];

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    __syncthreads();

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = warp.thread_rank(); i < C; i  += warp.size()) {
        float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = (float)weight[i] * (float)dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
    dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = (float)weight[i] * (float)dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias_shared[i], (float)dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight_shared[i], norm_bti * (float)dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] = (Tdinp)((float)dinp_bt[i] + dval);
    }
    __syncthreads();

    // write to global memory
    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        atomicAddX(&dbias[i], (Tparams)dbias_shared[i]);
        atomicAddX(&dweight[i], (Tparams)dweight_shared[i]);
    }
}


__global__ void softmax_autoregressive_backward_kernel(floatX* dpreatt, const floatX* datt, const floatX* att,
                                                       int B, int T, int C, float scale) {
    constexpr const int BlockSize = 256;
    constexpr int T_per_block = 4;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float block_acc[32];

    int idx = blockIdx.y;
    // go through blocks in reverse order, so the slowest block starts first
    int t0 = T - 1 - T_per_block*blockIdx.x;

    att += idx * T * T;
    datt += idx * T * T;
    dpreatt += idx * T * T;

    if (warp.meta_group_rank() == 0) {
        block_acc[warp.thread_rank()] = 0;
    }

    for(int to = 0; to < T_per_block; ++to) {
        int t = t0 - to;
        if(t < 0) return;
        const floatX* att_bth = att + t * T;
        const floatX* datt_bth = datt + t * T;
        floatX* dpreatt_bth = dpreatt + t * T;

        float local_sum = 0;
        for (int t2 = block.thread_rank(); t2 <= t; t2 += BlockSize) {
            local_sum += (float)att_bth[t2] * (float)datt_bth[t2];
        }

        block_acc[warp.meta_group_rank()] = cg::reduce(warp, local_sum, cg::plus<float>{});
        block.sync();
        local_sum = cg::reduce(warp, block_acc[warp.thread_rank()], cg::plus<float>{});

        for (int t3 = block.thread_rank(); t3 <= t; t3 += BlockSize) {
            // don't touch the cache. Some parts will still be here from the previous loop, and
            // we want to exploit those.
            float acc = (float)__ldcs(att_bth + t3) * ((float)__ldcs(datt_bth + t3) - local_sum);
            __stcs(dpreatt_bth + t3, (floatX)(scale * acc));
        }
    }
}

// Implements linear interpolation using only two floating-point operations (as opposed to three in a naive implementation).
// Reference: https://developer.nvidia.com/blog/lerp-faster-cuda
__device__ inline float lerp(float start, float end, float weight) {
    return fma(weight, end, fma(-weight, start, start));
}

// Termplate type T instead of floatx
template <typename Tp, typename Tg>
__global__ void adamw_kernel3(Tp* params_memory, Tg* grads_memory, float* m_memory, float* v_memory, size_t num_parameters,
                              float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay,
                              unsigned int seed) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= num_parameters) return;  // guard
   float grad = (float)grads_memory[i];
   float m = m_memory[i];
   float v = v_memory[i];
   // update the first moment (momentum)
   m = lerp(grad, m, beta1);
   m_memory[i] = m;
   // update the second moment (RMSprop)
   v = lerp(grad * grad, v, beta2);
   v_memory[i] = v;
   m /= beta1_correction;  // m_hat
   v /= beta2_correction;  // v_hat
   // update the parameters (weight/bias)
   float param = (float)params_memory[i] - (learning_rate * (m / (sqrtf(v) + eps) + weight_decay * (float)params_memory[i]));
   unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
   // todo - explain stochastic rounding here
   stochastic_rounding(param, &params_memory[i], random);
}

struct SoftmaxParams {
    float Scale;
    float Offset;
};

template <typename Type>
__device__ SoftmaxParams prepare_softmax_blockwide_nofloat4(cg::thread_block_tile<32>& warp,
                                                   int idx, const Type* inp, int V, int P) {
    // same but not float4
    // one row of inp, i.e. inp[idx, :] of shape (V,)

    const Type* x = inp + idx * P;
    float thread_maxval = -INFINITY;
    float thread_sumval = 0.0f;
    // do the loop in reverse to maximise probability of L2 cache hits
    // so even small L2s get some hits on the 2nd read of the same thread
    for (int i = V + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
        float v = (float)x[i];
        float old_maxval = thread_maxval;
        thread_maxval = fmaxf(thread_maxval, v);
        thread_sumval *= expf((old_maxval - thread_maxval));
        thread_sumval += expf(v - thread_maxval);
    }

    // two reductions of up to 1024 threads:
    // 1) inside warp (shuffle), 2) cross-warp (shared memory), 3) inside warp (shuffle)
    // this results in much cleaner assembly than a multi-warp cg::reduce
    __shared__ float shared_maxval[32];
    __shared__ float shared_sumval[32];
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    // reduce maxval within each warp
    float warp_maxval = cg::reduce(warp, thread_maxval, cg::greater<float>{});
    // thread 0 in each warp writes to shared memory
    if (lane_id == 0) { shared_maxval[warp_id] = warp_maxval; }
    __syncthreads();
    // each thread now loads the maxval across previous warps
    // if the thread is "out of range" of data, use -FLT_MAX as the maxval
    warp_maxval = (lane_id < num_warps) ? shared_maxval[lane_id] : -FLT_MAX;
    // now reduce the maxval among the warp threads
    float block_maxval = cg::reduce(warp, warp_maxval, cg::greater<float>{});
    // each thread uses maxval to scale sumval to avoid numerical instability / overflow
    thread_sumval *= expf(thread_maxval - block_maxval);
    // (warp-level) reduce sumval, thread 0 in each warp saves result in shared memory
    float warp_sumval = cg::reduce(warp, thread_sumval, cg::plus<float>{});
    if (lane_id == 0) { shared_sumval[warp_id] = warp_sumval; }
    __syncthreads();
    // same strategy, now reduce sumval across warps
    warp_sumval = (lane_id < num_warps) ? shared_sumval[lane_id] : 0.0f;
    float block_sumval = cg::reduce(warp, warp_sumval, cg::plus<float>{});
    // return the softmax parameters
    return SoftmaxParams{1.f / block_sumval, block_maxval};
}

// same as 2 but not using float4 (see dev/cuda/classifier_fused.cu)
// will _update_ logits to logit gradients
template <typename Type>
__global__ void fused_classifier_kernel3(Type* logits, Type* losses, Type* probs,
                                         const Type* dlosses, const int* targets,
                                         int B, int T, int V, int P) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x;
    int ix = targets[idx];

    // softmax (reading B * T * V, same logits read again below, hopefully still in cache)
    SoftmaxParams sp = prepare_softmax_blockwide_nofloat4(warp, idx, logits, V, P);

    // calculate the probability needed for the loss and update (single-threaded)
    if(threadIdx.x == 0) {
        float prob = expf((float)logits[idx * P + ix] - sp.Offset) * sp.Scale;
        losses[idx] = (Type)(-logf(prob));
    }

    // very sensible default for dlosses is 1/(B*T), which is the uniform loss
    float dloss = dlosses != NULL ? (float)dlosses[idx] : 1.0f / (B*T);
    // calculate the gradients directly, saves bandwidth from probs during training
    // but also supports writing probs for inference-only and debugging
    const Type* logits_vec = logits + idx * P;
    for (int i = threadIdx.x; i < V; i += blockDim.x) {
        // this is the 2nd read of logits after the one in prepare_softmax2
        // this data will never be needed again, so we reduce cache persistence
        float v = (float)__ldcs(&logits_vec[i]);
        float prob = expf(v - sp.Offset) * sp.Scale;
        if (probs != NULL) {
            probs[idx * P + i] = (Type)prob;
        }
        float indicator = (i == ix) ? 1.0f : 0.0f;
        logits[idx * P + i] = (Type)((prob - indicator) * dloss);
    }
}

// ----------------------------------------------------------------------------
// kernel launchers

template <typename TOut, typename Tw>
void encoder_forward(TOut* out,
                     int* inp, Tw* wte, Tw* wpe,
                     int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_forward_kernel2<<<grid_size, block_size>>>(out, inp, wte, wpe, B, T, C);
    cudaCheck(hipGetLastError());
}

template <typename Type, typename Tdout>
void encoder_backward(Type* dwte, Type* dwpe,
                    const Tdout* dout, const int* inp,
                    int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_backward_kernel<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

template <typename TOut, typename Type, typename Tparam>
void layernorm_forward(TOut* out, Type* mean, Type* rstd,
                       Type* inp, Tparam* weight, Tparam* bias,
                       int B, int T, int C) {
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// uses cuBLAS
void matmul_forward_cublas(floatX* out,
                    floatX* inp, floatX* weight, floatX* bias,
                    int B, int T, int C, int OC) {
    assert(bias == NULL); // bias is not supported for this kernel

    // FP16 alpha/beta need to be used if and only if HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;
    const half alpha_fp16 = (half)alpha, beta_fp16 = (half)beta;
    const void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&alpha_fp16 : (const void*)&alpha;
    const void* beta_ptr =  (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&beta_fp16 : (const void*)&beta;

    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B*T, C,
                             alpha_ptr, weight, CUBLAS_LOWP, C, inp, CUBLAS_LOWP, C, beta_ptr,
                             out, CUBLAS_LOWP, OC, CUBLAS_LOWP_COMPUTE, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// uses cuBLASLt to fuse the bias and gelu. does not work with OC = 50257 (last layer)
// https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu
void matmul_forward_cublaslt(floatX* out,
                     floatX* inp, floatX* weight, floatX* bias,
                     int B, int T, int C, int OC) {
    int has_bias = (bias != NULL);

    // check bias alignment
    if(((uintptr_t)bias % 16) != 0) {
        printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
        exit(EXIT_FAILURE);
    }

    // FP16 alpha/beta need to be used if and only if HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;
    const half alpha_fp16 = (half)alpha, beta_fp16 = (half)beta;
    const void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&alpha_fp16 : (const void*)&alpha;
    const void* beta_ptr =  (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&beta_fp16 : (const void*)&beta;

    int returnedResults = 0;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayout;
    hipblasLtMatrixLayout_t inputLayout;
    hipblasLtMatrixLayout_t outputLayout;
    hipblasLtMatrixLayout_t biasLayout;
    hipblasLtMatmulHeuristicResult_t heuristic;

    // create the operation descriptor
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_BIAS;

    hipDataType scale_type = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? HIP_R_16F : HIP_R_32F;
    cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, CUBLAS_LOWP_COMPUTE, scale_type));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    // define matrix layouts
    cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, CUBLAS_LOWP, C, OC, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, CUBLAS_LOWP, C, B*T, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, CUBLAS_LOWP, OC, B*T, OC));
    cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, CUBLAS_LOWP, OC, 1, OC));

    // create a preference handle with specified max workspace
    cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
    cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    // find a suitable algorithm
    cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
        weightLayout, inputLayout, outputLayout, outputLayout,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d\n", B, T, C, OC, has_bias);
        exit(EXIT_FAILURE);
    }

    // call the matmul
    cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
        alpha_ptr, weight, weightLayout, inp, inputLayout, beta_ptr,
        out, outputLayout, out, outputLayout, &heuristic.algo,
        cublaslt_workspace, cublaslt_workspace_size, 0));

    // cleanups
    cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
    cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
    cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}

void attention_forward(floatX* out, floatX* qkvr, floatX* att,
                       floatX* inp,
                       int B, int T, int C, int NH) {
    // Note: `inp` is not needed for backward pass, so we re-use it as a scratch buffer.
    // Its contents will be overwritten by this function.
    const int block_size = 256;
    const int softmax_block_size = 256;

    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    // IMPORTANT: alpha/beta are FP32 for HIPBLAS_COMPUTE_32F even if FP16 inputs/outputs
    // But need FP16 scale for HIPBLAS_COMPUTE_16F (no errors otherwise, just garbage results *sigh*)
    const float alpha = 1.0f;
    const float beta = 0.0f;
    const floatX alpha_lowp = (floatX)alpha;
    const floatX beta_lowp = (floatX)beta;
    void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? (void*)&alpha_lowp : (void*)&alpha;
    void* beta_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ? (void*)&beta_lowp : (void*)&beta;

    floatX* preatt = inp;
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS,
                                     alpha_ptr,
                                     k, CUBLAS_LOWP, HS, T * HS,
                                     q, CUBLAS_LOWP, HS, T * HS,
                                     beta_ptr,
                                     preatt, CUBLAS_LOWP, T, T * T,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int grid_size = CEIL_DIV(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);
    cudaCheck(hipGetLastError());

    // new approach: first cuBLAS another batched matmul
    floatX* vaccum = inp;
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T,
                                     alpha_ptr,
                                     v, CUBLAS_LOWP, HS, T * HS,
                                     att, CUBLAS_LOWP, T, T * T,
                                     beta_ptr,
                                     vaccum, CUBLAS_LOWP, HS, T * HS,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

template <typename TOut, typename T1, typename T2>
void residual_forward(TOut* out, T1* inp1, T2* inp2, int N) {
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    residual_forward_kernel<<<grid_size, block_size>>>(out, inp1, inp2, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward(floatX* out, const floatX* inp, int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_forward_kernel<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_backward(floatX* dinp, const floatX* inp, const floatX* dout, const int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_backward_kernel<<<grid_size, block_size>>>(dinp, inp, dout, N);
    cudaCheck(hipGetLastError());
}

void softmax_forward(float* out, float* inp, int N, int C) {
    int grid_size = N;
    const int block_size = 512;
    size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);
    softmax_forward_kernel7<<<grid_size, block_size, shared_mem_size>>>(out, inp, N, C);
    cudaCheck(hipGetLastError());
}

void matmul_backward(floatX* dinp, floatX* dweight, floatX* dbias,
                     floatX* dout, floatX* inp, floatX* weight,
                     int B, int T, int C, int OC) {
    float one = 1.0f;
    float zero = 0.0f;
    // backward to input, uses = in the backward pass (set the gradient)
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B*T, OC, &one,
                             weight, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &zero,
                             dinp, CUBLAS_LOWP, C, CUBLAS_LOWP_COMPUTE, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // backward to weight, uses += in the backward pass (accumulate the gradient)
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, B*T, &one,
                             inp, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &one,
                             dweight, CUBLAS_LOWP, C, CUBLAS_LOWP_COMPUTE, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // backward to bias, if given, does a +=
    if (dbias != NULL) {
        const int block_size = 1024;
        const int grid_size = OC / 32; // for now, OC must be divisible by 32 for this kernel to work
        matmul_backward_bias_kernel4<<<grid_size, block_size, block_size * sizeof(float)>>>(dbias, dout, B, T, OC);
        cudaCheck(hipGetLastError());
    }
}

template <typename Tdinp, typename Tparams, typename Tdout, typename Trest>
void layernorm_backward(Tdinp* dinp, Tparams* dweight, Tparams* dbias,
                        const Tdout* dout, const Trest* inp, const Tparams* weight, const Trest* mean, const Trest* rstd,
                        int B, int T, int C) {
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(32*N, block_size);
    size_t shared_mem_size = 2 * C * sizeof(float);
    layernorm_backward_kernel2<<<grid_size, block_size, shared_mem_size>>>(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);
    cudaCheck(hipGetLastError());
}

// the sequence of transformations in this compound op is:
// inp (B,T,3C) -> qkvr (B,T,3C) -> preatt (B,NH,T,T) -> att (B,NH,T,T) -> vaccum (B,T,C) -> out (B,T,C)
void attention_backward(floatX* dinp, floatX* dqkvr, floatX* dpreatt, floatX* datt, floatX* scratch,
                        const floatX* dout,
                        const floatX* qkvr, const floatX* att,
                        int B, int T, int C, int NH) {
    const int block_size = 256;
    int HS = C / NH; // head size

    // FP16 alpha/beta need to be used if and only if HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;
    const half alpha_fp16 = (half)alpha, beta_fp16 = (half)beta;
    const void* alpha_ptr = (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&alpha_fp16 : (const void*)&alpha;
    const void* beta_ptr =  (CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F) ?
                            (const void*)&beta_fp16 : (const void*)&beta;

    // unpack convenience pointers into q, k, v
    const floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    floatX *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;

    // backward through the unpermute operation
    int num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    cudaCheck(hipGetLastError());
    // backward into datt

    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, alpha_ptr,
                                           v, CUBLAS_LOWP, HS, T * HS, scratch, CUBLAS_LOWP, HS, T * HS, beta_ptr,
                                           datt, CUBLAS_LOWP, T, T * T, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));

    // backward into dv
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, alpha_ptr,
                                           scratch, CUBLAS_LOWP, HS, T * HS, att, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dv, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));

    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_autoregressive_backward_kernel<<<dim3(T / 4, B * NH), 256>>>(dpreatt, datt, att, B, T, C, scale);
    cudaCheck(hipGetLastError());
    // backward into q
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, alpha_ptr,
                                           k, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dq, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));
    // backward into k
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, alpha_ptr,
                                           q, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, beta_ptr,
                                           dk, CUBLAS_LOWP, HS, T * HS, B * NH, CUBLAS_LOWP_COMPUTE, HIPBLAS_GEMM_DEFAULT));
    // backward into inp
    num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
    permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

// replaces logits with logit gradients
template <typename Type>
void fused_classifier3(Type* logits, Type* losses,
                      const Type* dlosses, const int* targets,
                      int B, int T, int V, int P) {
    const int block_size = 1024;
    const int N = B * T;
    const int grid_size = N;
    fused_classifier_kernel3<<<grid_size, block_size>>>(logits, losses, (Type*)NULL, dlosses, targets, B, T, V, P);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
#define NUM_PARAMETER_TENSORS 16
typedef struct {
    floatX*   wte; // (V, C)
    floatX*   wpe; // (maxT, C)
    floatN*  ln1w; // (L, C)
    floatN*  ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatN*  ln2w; // (L, C)
    floatN*  ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatN*  lnfw; // (C)
    floatN*  lnfb; // (C)
} ParameterTensors;

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t V = config.vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = V * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // Set parameter sizes
    // floatN gives us an option to keep layernorm params in FP32 if we want to
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
    param_sizeof[2] = sizeof(floatN); // ln1w
    param_sizeof[3] = sizeof(floatN); // ln1b
    param_sizeof[8] = sizeof(floatN); // ln2w
    param_sizeof[9] = sizeof(floatN); // ln2b
    param_sizeof[14] = sizeof(floatN); // lnfw
    param_sizeof[15] = sizeof(floatN); // lnfb
}

// allocate memory for the parameters and point the individual tensors to the right places
float* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof, int on_device) {
    // calculate the number of parameters
    size_t num_parameters = 0;
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_elements[i];
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    // on_device: 0 = CPU, 1 = GPU
    float* params_memory;
    if (on_device) {
        cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    } else {
        params_memory = (float*)mallocCheck(num_parameters * sizeof(float)); // keep FP32 here
    }
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, (floatX**)&params->ln1w, (floatX**)&params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, (floatX**)&params->ln2w, (floatX**)&params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, (floatX**)&params->lnfw, (floatX**)&params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

#define NUM_ACTIVATION_TENSORS 21
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    floatX* ln1_mean; // (L, B, T)
    floatX* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    floatX* att; // (L, B, NH, T, T)
    floatX* attproj; // (L, B, T, C)
    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    floatX* ln2_mean; // (L, B, T)
    floatX* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* fcproj; // (L, B, T, C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C)
    floatX* lnf_mean; // (B, T)
    floatX* lnf_rstd; // (B, T)
    floatX* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;
} ActivationTensors;

void fill_in_activation_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t V = config.vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * C; // encoded
    act_sizes[1] = L * B * T * C; // ln1
    act_sizes[2] = L * B * T; // ln1_mean
    act_sizes[3] = L * B * T; // ln1_rstd
    act_sizes[4] = L * B * T * C; // atty
    act_sizes[5] = L * B * NH * T * T; // att
    act_sizes[6] = L * B * T * C; // attproj
    act_sizes[7] = L * B * T * C; // residual2
    act_sizes[8] = L * B * T * C; // ln2
    act_sizes[9] = L * B * T; // ln2_mean
    act_sizes[10] = L * B * T; // ln2_rstd
    act_sizes[11] = L * B * T * 4*C; // fch
    act_sizes[12] = L * B * T * 4*C; // fch_gelu
    act_sizes[13] = L * B * T * C; // fcproj
    act_sizes[14] = L * B * T * C; // residual3
    act_sizes[15] = B * T * C; // lnf
    act_sizes[16] = B * T; // lnf_mean
    act_sizes[17] = B * T; // lnf_rstd
    act_sizes[18] = B * T; // losses
    act_sizes[19] = L * B * T * 3*C; // qkvr
    act_sizes[20] = B * T * max(3*C, max(NH*T, V)); // output / scratch
}

// Backward pass is conceptually quite different from forward, because we can discard
// the activations of a layer as soon as we're done with it. This lets us aggressively
// reuse memory, so that we need far fewer tensors for backward state.
#define NUM_BACKWARD_TENSORS 3
typedef struct {
    floatX* bt4c; // (B, T, 4*C)
    floatX* preatt; // (B, NH, T, T)
    floatX* residual3; // (B, T, C)
} GradActTensors;

void fill_in_grad_act_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * 4 * C; // bt4c
    act_sizes[1] = B * NH * T * T; // preatt
    act_sizes[2] = B * T * C; // residual3
}

void* malloc_and_point(floatX** targets[], const size_t* act_sizes, size_t n) {
    size_t num_activations = 0;
    for (size_t i = 0; i < n; i++) {
        num_activations += act_sizes[i];
    }
    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(floatX)));
    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < n; i++) {
        *(targets[i]) = (floatX*)acts_memory_iterator;
        acts_memory_iterator += act_sizes[i] * sizeof(floatX);
    }
    return acts_memory;
}

void* malloc_and_point_activations(ActivationTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->atty,
        &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->losses, &acts->qkvr, &acts->output
    };
    return malloc_and_point(ptrs, act_sizes, NUM_ACTIVATION_TENSORS);
}

void* malloc_and_point_backward(GradActTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->bt4c, &acts->preatt, &acts->residual3
    };
    return malloc_and_point(ptrs, act_sizes, NUM_BACKWARD_TENSORS);
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    size_t num_activations;
    // gradients of the activations
    GradActTensors grads_acts;
    size_t num_grad_acts;
    void* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float accumulated_mean_loss; // Mean loss after aggregating it on all GPUs
    floatX* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
} GPT2;

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path) {

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file"); exit(EXIT_FAILURE); }
    if (model_header[1] != 1) { printf("Bad version in model file"); exit(EXIT_FAILURE); }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];

    // allocate space for all the parameters and read them in
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);

    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }
    size_t input_model_bytes = model->num_parameters * sizeof(float);

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof, 1);

    // read in all the parameters from file and copy them to device
    float* params_memory_cpu = (float*)mallocCheck(input_model_bytes);
    freadCheck(params_memory_cpu, 1, input_model_bytes, model_file);

    float* params_cpu_iterator = (float*)params_memory_cpu;
    char* params_gpu_iterator = (char*)model->params_memory;

    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        if (model->param_sizeof[i] == sizeof(float)) {
            cudaCheck(hipMemcpy(params_gpu_iterator, params_cpu_iterator, model->param_elements[i] * sizeof(float), hipMemcpyHostToDevice));
        } else {
            // TODO: Currently only support float or floatX (cannot mix and match FP16/BF16 etc...)
            assert(model->param_sizeof[i] == sizeof(floatX));
            floatX* conversion_scratchpad = (floatX*)mallocCheck(model->param_elements[i] * sizeof(floatX));
            for (size_t j = 0; j < model->param_elements[i]; j++) {
                conversion_scratchpad[j] = (floatX)params_cpu_iterator[j];
            }
            cudaCheck(hipMemcpy(params_gpu_iterator, conversion_scratchpad, model->param_elements[i] * sizeof(floatX), hipMemcpyHostToDevice));
            free(conversion_scratchpad);
        }
        params_cpu_iterator += model->param_elements[i];
        params_gpu_iterator += model->param_elements[i] * model->param_sizeof[i];
    }
    free(params_memory_cpu);
    fcloseCheck(model_file);

    // other inits
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f will designate no loss
    model->rng_state = 13371337;
}

void gpt2_forward(GPT2 *model, int* inputs, int* targets, size_t B, size_t T) {
    // targets are optional and could be NULL
    // in this function we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    size_t V = model->config.vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // validate inputs, all indices must be in the range [0, V)
    for(int i = 0; i < B * T; i++) {
        assert(0 <= inputs[i] && inputs[i] < V);
        if (targets != NULL) {
            assert(0 <= targets[i] && targets[i] < V);
        }
    }

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // allocate the space
        fill_in_activation_sizes(model->act_sizes, B, T, model->config);
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        model->num_activations = num_activations;
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        printf0("allocated %d MiB for activations\n", (int)round(num_activations * sizeof(floatX) / (1024 * 1024)));
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(floatX)));
    } else {
        // validate B,T is consistent with how we've allocated the memory before
        // in principle we could get more clever here in the future, for now this is safest
        if (B != model->batch_size || T != model->seq_len) {
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
            exit(EXIT_FAILURE);
        }
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    floatX* residual;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C); // encoding goes into residual[0]

    for (int l = 0; l < L; l++) {

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatN* l_ln1w = params.ln1w + l * C;
        floatN* l_ln1b = params.ln1b + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatN* l_ln2w = params.ln2w + l * C;
        floatN* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_att = acts.att + l * B * NH * T * T;
        floatX* l_attproj = acts.attproj + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        floatX* l_fcproj = acts.fcproj + l * B * T * C;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        floatX* scratch = (floatX*)acts.output;

        // now do the forward pass
        layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C);
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH);
        matmul_forward_cublaslt(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C);
        residual_forward(l_residual2, residual, l_attproj, B*T*C);
        layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        matmul_forward_cublaslt(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C);
        residual_forward(l_residual3, l_residual2, l_fcproj, B*T*C);
    }

    residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_forward(acts.lnf, acts.lnf_mean, acts.lnf_rstd, residual, params.lnfw, params.lnfb, B, T, C);
    matmul_forward_cublas(acts.output, acts.lnf, params.wte, NULL, B, T, C, V);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        // fused classifier: does the forward pass and first part of the backward pass
        // we're passing dlosses = NULL, which will default them to 1.0f/(B*T), i.e. uniform loss
        fused_classifier3(acts.output, acts.losses, (floatX*)NULL, model->targets, B, T, V, V);
        // for convenience also evaluate the mean loss (TODO re-think this compute+sync point)
        // move the (B,T) losses to CPU
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(floatX), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i=0; i<B*T; i++) { mean_loss += (float)(model->cpu_losses[i]); }
        mean_loss /= B*T;
        model->mean_loss = mean_loss;

    } else {
        // if we don't have targets, we don't have loss
        model->mean_loss = -1.0f;
    }
}

void gpt2_zero_grad(GPT2 *model) {
    if (model->grads_acts_memory != NULL) { cudaCheck(hipMemset(model->grads_acts_memory, 0, model->num_grad_acts * sizeof(floatX))); }
    if (model->grads_memory != NULL) { cudaCheck(hipMemset(model->grads_memory, 0, model->num_parameters * sizeof(floatX))); }
}

void gpt2_backward(GPT2 *model) {
    // double check we forwarded previously, with targets
    if (model->mean_loss == -1.0f) {
        printf("Error: must forward with targets before backward\n");
        exit(EXIT_FAILURE);
    }

    // lazily allocate the memory for gradients of the weights and activations, if needed
    if (model->grads_memory == NULL) {
        // allocate buffers for weight gradients
        model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof, 1);
        printf0("allocated %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
        // we're going to be clever for the activations backward pass. we don't need to exactly
        // mirror the forward pass acrtivations and we will save memory.
        size_t bw_act_sizes[NUM_ACTIVATION_TENSORS];
        GPT2Config cfg = model->config;
        cfg.num_layers = 1; // copy the configuration but override number of layers to 1
        fill_in_grad_act_sizes(bw_act_sizes, model->batch_size, model->seq_len, cfg);
        // count up and allocate the space
        model->grads_acts_memory = malloc_and_point_backward(&model->grads_acts, bw_act_sizes);
        model->num_grad_acts = 0;
        for (size_t i = 0; i < NUM_BACKWARD_TENSORS; i++) {
            model->num_grad_acts += bw_act_sizes[i];
        }
        printf0("allocated %d MiB for activation gradients\n", (int)round(model->num_grad_acts * sizeof(floatX) / (1024 * 1024)));
        // init gradients of parameters and activations to zero
        gpt2_zero_grad(model);
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    size_t B = model->batch_size;
    size_t T = model->seq_len;
    size_t V = model->config.vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // backward pass: go in the reverse order of the forward pass, and call backward() functions
    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;
    GradActTensors grads_acts = model->grads_acts;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(grads_acts.bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, B, T, C, V);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    floatX* dresidual = (floatX*)grads_acts.residual3; // the main buffer holding the gradient in the backward pass
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, grads_acts.bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C);

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatN* l_ln1w = params.ln1w + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatN* l_ln2w = params.ln2w + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatN* dl_ln1w = grads.ln1w + l * C;
        floatN* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatN* dl_ln2w = grads.ln2w + l * C;
        floatN* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_att = acts.att + l * B * NH * T * T;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        // we need a B x T x C buffer; thankfully, the forward activation for lnf isn't needed anymore,
        // so we can co-opt it here.
        floatX* dl_btc = (floatX*)acts.lnf;
        floatX* dl_bt4c = (floatX*)grads_acts.bt4c;
        floatX* dl_preatt = (floatX*)grads_acts.preatt;

        // re-use scratch buffer of the forward pass
        floatX* scratch = (floatX*)acts.output;

        // backprop this layer
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, B, T, 4*C, C);
        gelu_backward(dl_bt4c, l_fch, dl_bt4c, B*T*4*C);
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, B, T, C, 4 * C);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, B, T, C, C);
        // we more B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch;        // this is B x T x 4C, so even larger than what we need

        attention_backward(dl_bt4c, buffer_b, dl_preatt, scratch, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH);
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, B, T, C, 3 * C);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C);
    }
    encoder_backward(grads.wte, grads.wpe, dresidual, model->inputs, B, T, C);
}

// Compute a mean of a single CPU value across all GPU processes. No-op when multi-GPU is disabled.
float multi_gpu_cpu_float_mean(float value, const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    // MPI doesn't support all reduce with mean, so we sum up, then divide.
    float result;
    mpiCheck(MPI_Allreduce(&value, &result, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    return result / multi_gpu_config->num_processes;
#else
    return value;
#endif
}

// Averages out the loss and gradients across all GPUs. No-op when multi-GPU is disabled.
void gpt2_mutli_gpu_accumulate(GPT2* model, MultiGpuConfig* multi_gpu_config) {
    // Average all losses.
    model->accumulated_mean_loss = multi_gpu_cpu_float_mean(model->mean_loss, multi_gpu_config);
#ifdef MULTI_GPU
    // Average all gradients.
    char* grads_memory_iterator = (char*)model->grads_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; ++i) {
        int current_param_sizeof = model->param_sizeof[i];
        int current_param_elements = model->param_elements[i];
        ncclDataType_t data_type = current_param_sizeof == sizeof(floatX) ? ncclFloatX : ncclFloatN;
        ncclCheck(ncclAllReduce(grads_memory_iterator, grads_memory_iterator,
            current_param_elements,
            data_type, ncclAvg,
            multi_gpu_config->nccl_comm,
            // use 0 for default stream (all other computations use this stream)
            /*stream=*/0));
        grads_memory_iterator += current_param_elements * current_param_sizeof;
    }
    assert(grads_memory_iterator == (char*)model->grads_memory + model->num_parameters_bytes);
#endif
}

void gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, int t) {
    // reference: https://pytorch.org/docs/stable/generated/torch.optim.AdamW.html

    // lazily allocate the memory for m_memory and v_memory
    if (model->m_memory == NULL) {
        cudaCheck(hipMalloc((void**)&model->m_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->m_memory, 0, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, model->num_parameters * sizeof(float)));
        printf0("allocated %d MiB for AdamW optimizer state m\n", (int)round(model->num_parameters * sizeof(float) / (1024 * 1024)));
        printf0("allocated %d MiB for AdamW optimizer state v\n", (int)round(model->num_parameters * sizeof(float) / (1024 * 1024)));
    }

    int block_size = 512;
    float beta1_correction = 1.0f - powf(beta1, t);
    float beta2_correction = 1.0f - powf(beta2, t);

    // Do adam per set of parameters
    // We need to know the parameter types (float or floatX) to process consecutive chunks
    // TODO - optimise this to require fewer kernel launches and/or independent via CUDA streams
    char* params_mem = (char*)model->params_memory;
    char* grads_mem = (char*)model->grads_memory;
    size_t num_elements = model->param_elements[0];
    size_t last_sizeof = model->param_sizeof[0];
    size_t current_element = 0;

    for (int i = 1; i <= NUM_PARAMETER_TENSORS; i++) {
        if (i == NUM_PARAMETER_TENSORS || model->param_sizeof[i] != last_sizeof) {
            unsigned int seed = random_u32(&model->rng_state); // seed for stochastic rounding
            int num_blocks = CEIL_DIV(num_elements, block_size);
            // atm some params are in low precision (floatX) and some are in high precision (float)
            if (last_sizeof == sizeof(floatX)) {
                adamw_kernel3<<<num_blocks, block_size>>>((floatX*)params_mem, (floatX*)grads_mem,
                            &model->m_memory[current_element], &model->v_memory[current_element], num_elements,
                            learning_rate, beta1, beta2, beta1_correction, beta2_correction, eps, weight_decay, seed);
            } else {
                adamw_kernel3<<<num_blocks, block_size>>>((float*)params_mem, (float*)grads_mem,
                            &model->m_memory[current_element], &model->v_memory[current_element], num_elements,
                            learning_rate, beta1, beta2, beta1_correction, beta2_correction, eps, weight_decay, seed);
            }
            params_mem += num_elements * last_sizeof;
            grads_mem += num_elements * last_sizeof;
            current_element += num_elements;
            num_elements = 0;
        }
        if (i != NUM_PARAMETER_TENSORS) {
            num_elements += model->param_elements[i];
            last_sizeof = model->param_sizeof[i];
        }
    }
    cudaCheck(hipGetLastError());
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    hipHostFree(model->cpu_losses);
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip the int main below

// ----------------------------------------------------------------------------
// data loader lite: returns random batches of data from a file of integers

typedef struct {
    // Distributed data parallel specifics.
    // Each worker loads it's own chunk of data.
    int process_rank;
    int num_processes;
    // hyperparameters. use size_t to prevent overflow
    size_t B;
    size_t T;
    // input handling and its state
    FILE* tokens_file;
    long file_size;
    long current_position;
    // output memory
    int* batch;
    int* inputs;
    int* targets;
    // convenience variables
    size_t num_batches;
} DataLoader;

void dataloader_init(DataLoader *loader, const MultiGpuConfig* multi_gpu_config, const char* filename, size_t B, size_t T) {
    loader->process_rank = multi_gpu_config->process_rank;
    loader->num_processes = multi_gpu_config->num_processes;
    loader->B = B;
    loader->T = T;

    // open the input file for reading
    loader->tokens_file = fopenCheck(filename, "rb");

    // determine the file size
    fseek(loader->tokens_file, 0, SEEK_END);
    loader->file_size = ftell(loader->tokens_file);
    fseek(loader->tokens_file, 0, SEEK_SET);
    if (loader->file_size < (B * T + 1) * sizeof(int)) {
        printf("Error: file size is too small for the batch size and sequence length\n");
        exit(EXIT_FAILURE);
    }
    loader->current_position = loader->process_rank * B * T * sizeof(int); // start at the beginning

    // allocate space for B*T + 1 integers to store the inputs and targets
    // Using CUDA CPU pinned memory for faster PCI Express transfers to GPU
    // See: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
    hipHostMalloc((void**)&loader->batch, (B * T + 1) * sizeof(int));
    loader->inputs = loader->batch;
    loader->targets = loader->batch + 1; // targets are shifted by one
    loader->num_batches = loader->file_size / (loader->num_processes * B * T * sizeof(int));
}

void dataloader_reset(DataLoader *loader) {
    loader->current_position = 0;
}

void dataloader_next_batch(DataLoader *loader) {
    size_t B = loader->B;
    size_t T = loader->T;
    // if we are at the end of the file, loop back to the beginning
    if (loader->current_position + (loader->num_processes * B * T + 1) * sizeof(int) > loader->file_size) {
        loader->current_position = loader->process_rank * B * T * sizeof(int);
    }
    // read the B*T+1 integers from the file into batch
    fseek(loader->tokens_file, loader->current_position, SEEK_SET);
    freadCheck(loader->batch, sizeof(int), B*T+1, loader->tokens_file);
    // advance the current position by B*T*num_processes integers
    loader->current_position += loader->num_processes * B * T * sizeof(int);
}

void dataloader_free(DataLoader *loader) {
    fcloseCheck(loader->tokens_file);
    hipHostFree(loader->batch);
}

// ----------------------------------------------------------------------------
// sampler: takes probabilities and samples integers from them

#define GPT2_EOT 50256

int sample_softmax(const float* logits, int n, float coin) {
    // sample index from logits (converted to probabilities using softmax)
    // coin is a random number in [0, 1), usually from random_f32()
    double norm = 0;
    for (int i = 0; i < n; i++) {
        norm += expf(logits[i]);
    }
    // instead of dividing all exp(logits), we can just multiply coin.
    coin *= norm;
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += expf(logits[i]);
        if (coin < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

// ----------------------------------------------------------------------------
// Tokenizer (only supports decoding: tokens (integers) -> strings)

typedef struct {
    uint32_t vocab_size;
    char **token_table;
    int init_ok;
} Tokenizer;

void safe_printf(const char *piece) {
    // the tokens are raw bytes, and we we only want to print the printable ones
    // many bytes can be various control codes, backspace, etc.
    if (piece == NULL) { return; }
    if (piece[0] == '\0') { return; }
    // handle individual byte tokens
    // every token is asserted to be at least one byte so doing piece[1] is ok
    if (piece[1] == '\0') {
        unsigned char byte_val = piece[0];
        if (!(isprint(byte_val) || isspace(byte_val))) {
            return; // weird byte, don't print it
        }
    }
    printf("%s", piece);
}

void tokenizer_init(Tokenizer *tokenizer, const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        // try to be more helpful as we just added this feature, erase later
        printf("---\n");
        printf("WARNING: Failed to open the tokenizer file %s\n", filename);
        printf("The Tokenizer is a new feature added April 14 2024.\n");
        printf("Re-run `python train_gpt2.py` to write it\n");
        printf("---\n");
        tokenizer->init_ok = 0;
        return;
    }
    // read in the header
    uint32_t header[256];
    freadCheck(header, sizeof(uint32_t), 256, file);
    assert(header[0] == 20240328);
    assert(header[1] == 1);
    tokenizer->vocab_size = header[2];
    // read in all the tokens
    unsigned char length;
    tokenizer->token_table = (char **)mallocCheck(tokenizer->vocab_size * sizeof(char *));
    for (uint32_t i = 0; i < tokenizer->vocab_size; i++) {
        freadCheck(&length, sizeof(unsigned char), 1, file);
        assert(length > 0); // every token should be at least one character
        char *token_bytes = (char *)mallocCheck(length + 1);
        freadCheck(token_bytes, sizeof(char), length, file);
        token_bytes[length] = '\0';  // Add null terminator for printing
        tokenizer->token_table[i] = token_bytes;
    }
    // cleanups
    fcloseCheck(file);
    tokenizer->init_ok = 1;
}

const char *tokenizer_decode(Tokenizer *tokenizer, uint32_t token_id) {
    if (tokenizer->init_ok == 0) {
        return NULL;
    }
    if (token_id < tokenizer->vocab_size) {
        return tokenizer->token_table[token_id];
    } else {
        printf("invalid token id %d!\n", token_id);
        return NULL;
    }
}

void tokenizer_free(Tokenizer *tokenizer) {
    if (tokenizer->init_ok) {
        for (uint32_t i = 0; i < tokenizer->vocab_size; i++) {
            free(tokenizer->token_table[i]);
        }
        free(tokenizer->token_table);
    }
}

// ----------------------------------------------------------------------------
// Logger lite, will probably grow/change some over time

typedef struct {
    FILE *logfile;
    int flush_every; // every how many steps to flush the log
} Logger;

void logger_init(Logger *logger, const char *filename) {
    logger->flush_every = 20;
    logger->logfile = NULL;
    if (filename != NULL) { logger->logfile = fopenCheck(filename, "w"); }
}

void logger_log_val(Logger *logger, int step, float val_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d tel:%.4f\n", step, val_loss);
    }
}

void logger_log_train(Logger *logger, int step, float train_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d trl:%.4f\n", step, train_loss);
        if (step % 10 == 0) { fflush(logger->logfile); }
    }
}

void logger_free(Logger *logger) {
    if (logger->logfile != NULL) { fclose(logger->logfile); }
}

// ----------------------------------------------------------------------------
// CLI, poor man's argparse

void error_usage() {
    // default run = debugging run with TinyShakespeare
    // bigger run = train on TinyStories! e.g. val/sample less often, but sample more tokens, write to logfile
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Example: ./train_gpt2cu -i data/TinyStories -v 100 -s 100 -g 144 -o stories.log\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -i <string> input dataset prefix (default = data/tiny_shakespeare)\n");
    fprintf(stderr, "  -o <string> output log file (default = NULL)\n");
    fprintf(stderr, "  -b <int>    batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_batches, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    multi_gpu_config = multi_gpu_config_init(&argc, &argv);

    // read in the (optional) command line arguments
    const char* input_dataset_prefix = "data/tiny_shakespeare"; // or e.g. data/TinyStories
    const char* output_log_file = NULL;
    int B = 4; // batch size
    int T = 1024; // sequence length max
    float learning_rate = 3e-4f;
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_batches = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 'i') { input_dataset_prefix = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_file = argv[i+1]; }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_batches = atoi(argv[i+1]); }
        else if (argv[i][1] == 's') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else { error_usage(); }
    }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| input dataset prefix  | %-50s |\n", input_dataset_prefix);
    printf0("| output log file       | %-50s |\n", output_log_file == NULL ? "NULL" : output_log_file);
    printf0("| batch size B          | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| learning rate         | %-50f |\n", learning_rate);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_batches       | %-50d |\n", val_max_batches);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // set up the device
    cudaCheck(hipSetDevice(multi_gpu_config.local_device_idx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx);
    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));
    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| TF32                  | %-50s |\n", enable_tf32 ? "enabled" : "disabled");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    char train_tokens_filename[128];
    char val_tokens_filename[128];
    assert(strlen(input_dataset_prefix) < 100); // being bit lazy here, make sure we don't overflow
    sprintf(train_tokens_filename, "%s_train.bin", input_dataset_prefix);
    sprintf(val_tokens_filename, "%s_val.bin", input_dataset_prefix);
    DataLoader train_loader;
    dataloader_init(&train_loader, &multi_gpu_config, train_tokens_filename, B, T);
    DataLoader val_loader;
    dataloader_init(&val_loader, &multi_gpu_config, val_tokens_filename, B, T);
    int train_num_batches = train_loader.num_batches; // let's do 1 epoch by default for now
    int val_num_batches = train_loader.num_batches < val_max_batches ? train_loader.num_batches : val_max_batches;
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // more prints related to allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu ==> bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));

    // set up the Logger
    Logger logger;
    logger_init(&logger, output_log_file);

    // build the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // some memory for generating samples from the model
    unsigned long long rng_state = 1337;
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // train
    struct timespec start, end;
    double total_sum_iteration_time_s = 0.0;
    for (int step = 0; step <= train_num_batches; step++) {
        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss
        if (step % val_loss_every == 0 || last_step) {
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_mean(val_loss, &multi_gpu_config);
            printf("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while do model inference to print generated text
        if (multi_gpu_config.process_rank == 0 && (step > 0 && (step % sample_every) == 0 || last_step)) {
            // fill up gen_tokens with the GPT2_EOT, which kicks off the generation
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = GPT2_EOT;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                // note that inference is very wasteful here because for each token
                // we re-calculate the forward pass for all of (B,T) positions from scratch
                // but the inference here is just for sanity checking anyway
                // and we can maybe optimize a bit more later, with careful tests
                gpt2_forward(&model, gen_tokens, NULL, B, T);
                // furthermore, below we're only using b=0 (i.e. the first row) of all B rows
                // we're in principle running B "inference streams" in parallel here
                // only using position 0 because it's a bit faster (copy less probs from GPU -> CPU)
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.vocab_size;
                // move probs back to CPU and sample
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }

                float coin = random_f32(&rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // do a training step
        clock_gettime(CLOCK_MONOTONIC, &start);
        dataloader_next_batch(&train_loader);
        gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T);
        gpt2_zero_grad(&model);
        gpt2_backward(&model);
        gpt2_mutli_gpu_accumulate(&model, &multi_gpu_config);
        gpt2_update(&model, learning_rate, 0.9f, 0.999f, 1e-8f, 0.0f, step+1);
        cudaCheck(hipDeviceSynchronize()); // finish all CUDA work to get correct precise timings
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
        total_sum_iteration_time_s += time_elapsed_s;
        int tokens_per_second = multi_gpu_config.num_processes * (B * T) / time_elapsed_s;
        printf0("step %4d/%d: train loss %f (acc %f) (%f ms, %d tok/s)\n", step + 1, train_num_batches, model.mean_loss, model.accumulated_mean_loss, time_elapsed_s * 1000, tokens_per_second);
        logger_log_train(&logger, step, model.mean_loss);
    }
    // add a total average, for optimizations that are only mild improvements
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / train_num_batches * 1000);

    // free
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    gpt2_free(&model);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    logger_free(&logger);
    multi_gpu_config_free(&multi_gpu_config);

    return 0;
}
#endif
